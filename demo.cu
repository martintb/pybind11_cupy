#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
namespace py = pybind11;

// Error checking macro
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// CUDA kernel function to add the elements of two arrays
__global__
void cuadd(int n, float *x, float *y, float *z)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    z[i] = x[i] + y[i];
}

// Function to be called by Python
void pyadd(int N, size_t px, size_t py, size_t pz)
{

  float *x = reinterpret_cast<float*> (px);
  float *y = reinterpret_cast<float*> (py);
  float *z = reinterpret_cast<float*> (pz);

  // Run kernel on GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  cuadd<<<numBlocks, blockSize>>>(N,x,y,z);

  // Wait for GPU to finish before accessing on host
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
}

// This is the code to expose the pyadd function to Python
PYBIND11_MODULE(demo, m) {
  m.doc() = "pybind11 example plugin"; // optional module docstring
  m.def("pyadd", &pyadd, "A function which adds two cupy arrays");
}

